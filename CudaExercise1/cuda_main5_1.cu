// 이번엔 2차원 행렬 곱에 대해 gpu연산을 수행해서 cpu 대비 시간을 측정해보자
// cpu 행렬곱 연산 코드가 너무 지저분해서 수정한 코드


#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
#include <iostream>

// ns 단위의 시간 측정을 위해
#include <chrono>

// matrix 계산을 위해
#include <vector>
// vector 쓰지말고 그냥 직접 메모리 주소로 접근하자

using namespace std;

// cpu 
int *MakeSquareMatrix(int matrix_size) {
	// 2차원 벡터 생성
	int *a = new int[matrix_size * matrix_size];

	// 2차원 벡터 초기화
	for (int i = 0; i < matrix_size*matrix_size; i++) {
		*(a + i) = i + 1;
	}
	cout << "a의 주소값 : " << a << endl;
	return a;
}

int *MatrixMul(int* matrix_A, int* matrix_B, int _matrix_size) {			// 2차원 vector 같은거 쓰지 않고 포인터로 입력받고 출력했다.
	int matrix_size = _matrix_size;
	int *matrix_C = new int[matrix_size*matrix_size];
	for (int i = 0; i < matrix_size*matrix_size; i++) {
		*(matrix_C + i) = 0;
	}

	int temp1, temp2;
	for (int i = 0; i < matrix_size; i++) {
		for (int j = 0; j < matrix_size; j++) {
			int sum = 0;
			for (int k = 0; k < matrix_size; k++) {
				//sum += (matrix_A[i][k] * matrix_B[k][j]);
				sum += (*(matrix_A + (i*matrix_size) + k) * *(matrix_B + (k*matrix_size) + j));
			}
			//matrix_C[i][j] = sum;
			*(matrix_C + (i*matrix_size)+j) = sum;
		}
	}

	return matrix_C;
}

void PrintMatrix(int* matrix, int _matrix_size) {
	// 2차원 벡터 출력
	int matrix_size = _matrix_size;
	for (int i = 0; i < matrix_size * matrix_size; i++) {
		cout << *(matrix + i) << " ";
		if ((i + 1) % _matrix_size == 0)
			cout << endl;
	}
}



// gpu

__global__ void KernelMatrixMul(int* matrix_A, int* matrix_B, int* matrix_C, int matrix_size) {
	int ROW = blockIdx.y*blockDim.y + threadIdx.y;
	int COL = blockIdx.x*blockDim.x + threadIdx.x;

	if (ROW < matrix_size && COL < matrix_size) {
		int temp = 0;
		for (int i = 0; i < matrix_size; i++) {
			temp = matrix_A[ROW*matrix_size + i] * matrix_B[COL*matrix_size + i];
		}
		matrix_C[ROW*matrix_size + COL] = temp;
	}
}


int main(void) {
	// cpu
	// 우선 cpu를 이용한 행렬 곱 연산에 대해 구현합니다.
	// 시간이 얼마나 걸릴지 확인해봅시다.
	int matrix_size = 30;
	int* matrix_A = new int[matrix_size * matrix_size];
	int* matrix_B = new int[matrix_size * matrix_size];
	int* matrix_C = new int[matrix_size * matrix_size];
	matrix_A = MakeSquareMatrix(matrix_size);
	matrix_B = MakeSquareMatrix(matrix_size);

	auto start_cpu = std::chrono::high_resolution_clock::now();
	matrix_C = MatrixMul(matrix_A, matrix_B, matrix_size);
	auto end_cpu = std::chrono::high_resolution_clock::now();
	int result_cpu = (int)(end_cpu - start_cpu).count();
	int* matrix_C_print = new int[matrix_size * matrix_size];
	matrix_C_print = matrix_C;
	//cout << " Matrix A" << endl;
	//PrintMatrix(matrix_A, matrix_size);
	//cout << " Matrix B" << endl;
	//PrintMatrix(matrix_B, matrix_size);
	//cout << " Matrix C" << endl;
	//PrintMatrix(matrix_C, matrix_size);

	// gpu
	int *dev_matrix_A = new int[matrix_size*matrix_size];
	int *dev_matrix_B = new int[matrix_size*matrix_size];
	int *dev_matrix_C = new int[matrix_size*matrix_size];

	hipMemcpy(dev_matrix_A, matrix_A, matrix_size*matrix_size, hipMemcpyHostToDevice);
	hipMemcpy(dev_matrix_B, matrix_B, matrix_size*matrix_size, hipMemcpyHostToDevice);

	auto start_gpu = std::chrono::high_resolution_clock::now();
	KernelMatrixMul <<<matrix_size, matrix_size >>> (dev_matrix_A, dev_matrix_B, dev_matrix_C, matrix_size);
	auto end_gpu = std::chrono::high_resolution_clock::now();
	int result_gpu = (int)(end_gpu - start_gpu).count();

	hipMemcpy(matrix_C, dev_matrix_C, matrix_size*matrix_size, hipMemcpyDeviceToHost);

	std::cout << "cpu 시간	gpu 시간" << std::endl;
	std::cout << result_cpu << "ns	" << result_gpu << "ns" << std::endl;
	std::cout << std::endl;
	std::cout << "gpu가 cpu보다 " << ((double)result_cpu / result_gpu) << "배 빠름" << std::endl;

	PrintMatrix(matrix_C_print, matrix_size);
	PrintMatrix(matrix_C, matrix_size);

	system("pause");
	return 0;
}