// �̹��� 2���� ��� ���� ���� gpu������ �����ؼ� cpu ��� �ð��� �����غ���
// cpu ��İ� ���� �ڵ尡 �ʹ� �������ؼ� ������ �ڵ�


#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
#include <iostream>

// ns ������ �ð� ������ ����
#include <chrono>

// matrix ����� ����
#include <vector>
// vector �������� �׳� ���� �޸� �ּҷ� ��������

using namespace std;

// cpu 
int *MakeSquareMatrix(int matrix_size) {
	// 2���� ���� ����
	int *a = new int[matrix_size * matrix_size];

	// 2���� ���� �ʱ�ȭ
	for (int i = 0; i < matrix_size*matrix_size; i++) {
		*(a + i) = i + 1;
	}
	cout << "a�� �ּҰ� : " << a << endl;
	return a;
}

int *MatrixMul(int* matrix_A, int* matrix_B, int _matrix_size) {			// 2���� vector ������ ���� �ʰ� �����ͷ� �Է¹ް� ����ߴ�.
	int matrix_size = _matrix_size;
	int *matrix_C = new int[matrix_size*matrix_size];
	for (int i = 0; i < matrix_size*matrix_size; i++) {
		*(matrix_C + i) = 0;
	}

	int temp1, temp2;
	for (int i = 0; i < matrix_size; i++) {
		for (int j = 0; j < matrix_size; j++) {
			int sum = 0;
			for (int k = 0; k < matrix_size; k++) {
				//sum += (matrix_A[i][k] * matrix_B[k][j]);
				sum += (*(matrix_A + (i*matrix_size) + k) * *(matrix_B + (k*matrix_size) + j));
			}
			//matrix_C[i][j] = sum;
			*(matrix_C + (i*matrix_size)+j) = sum;
		}
	}

	return matrix_C;
}

void PrintMatrix(int* matrix, int _matrix_size) {
	// 2���� ���� ���
	int matrix_size = _matrix_size;
	for (int i = 0; i < matrix_size * matrix_size; i++) {
		cout << *(matrix + i) << " ";
		if ((i + 1) % _matrix_size == 0)
			cout << endl;
	}
}



// gpu

__global__ void KernelMatrixMul(int* matrix_A, int* matrix_B, int* matrix_C, int matrix_size) {
	int ROW = blockIdx.y*blockDim.y + threadIdx.y;
	int COL = blockIdx.x*blockDim.x + threadIdx.x;

	if (ROW < matrix_size && COL < matrix_size) {
		int temp = 0;
		for (int i = 0; i < matrix_size; i++) {
			temp = matrix_A[ROW*matrix_size + i] * matrix_B[COL*matrix_size + i];
		}
		matrix_C[ROW*matrix_size + COL] = temp;
	}
}


int main(void) {
	// cpu
	// �켱 cpu�� �̿��� ��� �� ���꿡 ���� �����մϴ�.
	// �ð��� �󸶳� �ɸ��� Ȯ���غ��ô�.
	int matrix_size = 800;
	int* matrix_A = new int[matrix_size * matrix_size];
	int* matrix_B = new int[matrix_size * matrix_size];
	int* matrix_C = new int[matrix_size * matrix_size];
	matrix_A = MakeSquareMatrix(matrix_size);
	matrix_B = MakeSquareMatrix(matrix_size);

	auto start_cpu = std::chrono::high_resolution_clock::now();
	matrix_C = MatrixMul(matrix_A, matrix_B, matrix_size);
	auto end_cpu = std::chrono::high_resolution_clock::now();
	int result_cpu = (int)(end_cpu - start_cpu).count();
	//cout << " Matrix A" << endl;
	//PrintMatrix(matrix_A, matrix_size);
	//cout << " Matrix B" << endl;
	//PrintMatrix(matrix_B, matrix_size);
	//cout << " Matrix C" << endl;
	//PrintMatrix(matrix_C, matrix_size);

	// gpu
	int *dev_matrix_A = new int[matrix_size*matrix_size];
	int *dev_matrix_B = new int[matrix_size*matrix_size];
	int *dev_matrix_C = new int[matrix_size*matrix_size];

	hipMemcpy(dev_matrix_A, matrix_A, matrix_size*matrix_size, hipMemcpyHostToDevice);
	hipMemcpy(dev_matrix_B, matrix_B, matrix_size*matrix_size, hipMemcpyHostToDevice);

	auto start_gpu = std::chrono::high_resolution_clock::now();
	KernelMatrixMul <<<matrix_size, matrix_size >>> (dev_matrix_A, dev_matrix_B, dev_matrix_C, matrix_size);
	auto end_gpu = std::chrono::high_resolution_clock::now();
	int result_gpu = (int)(end_gpu - start_gpu).count();

	hipMemcpy(matrix_C, dev_matrix_C, matrix_size*matrix_size, hipMemcpyDeviceToHost);

	std::cout << "cpu �ð�	gpu �ð�" << std::endl;
	std::cout << result_cpu << "ns	" << result_gpu << "ns" << std::endl;
	std::cout << std::endl;
	std::cout << "gpu�� cpu���� " << ((double)result_cpu / result_gpu) << "�� ����" << std::endl;

	system("pause");
	return 0;
}