// cpu�� gpu �ð� ���� �� �غ���
// �ڵ忡 #if�� �����ߴµ�, ����� �̷��� �ڵ� ¥�� ����.
// ����� ���� �Ұ����ϰ� �Դ���������.


#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
#include <iostream>

#define SIZE 1024 * 1024 * 10
#define DEBUG 0
#define TIME_INSTANCE 0

#if TIME_INSTANCE
// �ð� ������ ���� �������
#include <time.h>
#else
// time.h�� ms ������ ������ִµ� �ʹ� ������ Ŀ�� �̼��� �ð��� �� �𸣰ڴ�.
#include <chrono>
#endif

__global__ void kernel_add(int *a, int *b, int *c) {
	//*c = *a + *b;
	// (###)
	c[blockIdx.x] = a[blockIdx.x] + b[blockIdx.x];
#if DEBUG
	printf("%d + %d = %d\n", a[blockIdx.x], b[blockIdx.x], c[blockIdx.x]);
#endif
}

void host_add(int *a, int *b, int *c) {
	for (int i = 0; i < SIZE; i++) {
		c[i] = a[i] + b[i];
	}
}

int main(void) {
#if TIME_INSTANCE
	clock_t start_cpu, end_cpu;
	clock_t start_gpu, end_gpu;
	double result_cpu, result_gpu;
#endif


	int *a = new int[SIZE];
	int *b = new int[SIZE];
	int *c = new int[SIZE];
	int *cpu_a = new int[SIZE];
	int *cpu_b = new int[SIZE];
	int *cpu_c = new int[SIZE];

	int *d_a, *d_b, *d_c;

	hipMalloc((void**)&d_a, sizeof(int) * SIZE);
	hipMalloc((void**)&d_b, sizeof(int) * SIZE);
	hipMalloc((void**)&d_c, sizeof(int) * SIZE);

	for (int i = 0; i < SIZE; i++) {
		a[i] = i;
		b[i] = i;
		cpu_a[i] = i;
		cpu_b[i] = i;
#if DEBUG
		std::cout << a[i] << " " << b[i] << std::endl;
#endif
	}

	// cpu
#if TIME_INSTANCE
	start_cpu = clock();
#else
	auto start_cpu = std::chrono::high_resolution_clock::now();
#endif
	host_add(cpu_a, cpu_b, cpu_c);
#if TIME_INSTANCE
	end_cpu = clock();
	result_cpu = (double)(end_cpu - start_cpu);
#else
	auto end_cpu = std::chrono::high_resolution_clock::now();
	int result_cpu = (int)(end_cpu - start_cpu).count();
#endif

	// gpu
	hipMemcpy(d_a, a, sizeof(int) * SIZE, hipMemcpyHostToDevice);
	hipMemcpy(d_b, b, sizeof(int) * SIZE, hipMemcpyHostToDevice);

#if TIME_INSTANCE
	start_gpu = clock();
#else
	auto start_gpu = std::chrono::high_resolution_clock::now();
#endif
	kernel_add <<<SIZE, 1 >>> (d_a, d_b, d_c);
#if TIME_INSTANCE
	end_gpu = clock();
	result_gpu = (double)(end_gpu - start_gpu);
#else
	auto end_gpu = std::chrono::high_resolution_clock::now();
	int result_gpu = (int)(end_gpu - start_gpu).count();
#endif

	hipMemcpy(c, d_c, sizeof(int) * SIZE, hipMemcpyDeviceToHost);

#if DEBUG
	// ��� ����� ����մϴ�.
	std::cout << "a b c" << std::endl;
	std::cout << a << " " << b << " " << c << std::endl;
	//std::cout << *a << " " << *b << " " << *c << std::endl;
	//std::cout << &a << " " << &b << " " << &c << std::endl;
	for (int i = 0; i < SIZE; ++i)
		printf("c[%d] = %d\n", i, c[i]);
#endif

	// ��� ��� Ȯ��
	for (int i = 0; i < SIZE; i++) {
		if (cpu_c[i] != c[i]) {
			std::cout << " ��� ����Ȯ�� " << i << " Ȯ�� �ʿ� "  << std::endl;
			break;
		}
	}

	std::cout << "### ��� ��Ȯ�� ###" << std::endl;
	std::cout << "cpu �ð�	gpu �ð�" << std::endl;
	std::cout << result_cpu << "ns	" << result_gpu << "ns" << std::endl;
	std::cout << std::endl;
	std::cout << "gpu�� cpu���� " << (result_cpu / result_gpu) << "�� ����" << std::endl;

	delete[] a;
	delete[] b;
	delete[] c;

	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_c);

	system("pause");

	return 0;

}