// cpu와 gpu 시간 차이 비교 해보기
// 코드에 #if를 남발했는데, 절대로 이렇게 코드 짜지 말자.
// 기록을 위해 불가피하게 쌉더러워졌다.


#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
#include <iostream>

#define SIZE 1024 * 1024 * 10
#define DEBUG 0
#define TIME_INSTANCE 0

#if TIME_INSTANCE
// 시간 측정을 위한 헤더파일
#include <time.h>
#else
// time.h는 ms 단위로 계산해주는데 너무 단위가 커서 미세한 시간을 잘 모르겠다.
#include <chrono>
#endif

__global__ void kernel_add(int *a, int *b, int *c) {
	//*c = *a + *b;
	// (###)
	c[blockIdx.x] = a[blockIdx.x] + b[blockIdx.x];
#if DEBUG
	printf("%d + %d = %d\n", a[blockIdx.x], b[blockIdx.x], c[blockIdx.x]);
#endif
}

void host_add(int *a, int *b, int *c) {
	for (int i = 0; i < SIZE; i++) {
		c[i] = a[i] + b[i];
	}
}

int main(void) {
#if TIME_INSTANCE
	clock_t start_cpu, end_cpu;
	clock_t start_gpu, end_gpu;
	double result_cpu, result_gpu;
#endif


	int *a = new int[SIZE];
	int *b = new int[SIZE];
	int *c = new int[SIZE];
	int *cpu_a = new int[SIZE];
	int *cpu_b = new int[SIZE];
	int *cpu_c = new int[SIZE];

	int *d_a, *d_b, *d_c;

	hipMalloc((void**)&d_a, sizeof(int) * SIZE);
	hipMalloc((void**)&d_b, sizeof(int) * SIZE);
	hipMalloc((void**)&d_c, sizeof(int) * SIZE);

	for (int i = 0; i < SIZE; i++) {
		a[i] = i;
		b[i] = i;
		cpu_a[i] = i;
		cpu_b[i] = i;
#if DEBUG
		std::cout << a[i] << " " << b[i] << std::endl;
#endif
	}

	// cpu
#if TIME_INSTANCE
	start_cpu = clock();
#else
	auto start_cpu = std::chrono::high_resolution_clock::now();
#endif
	host_add(cpu_a, cpu_b, cpu_c);
#if TIME_INSTANCE
	end_cpu = clock();
	result_cpu = (double)(end_cpu - start_cpu);
#else
	auto end_cpu = std::chrono::high_resolution_clock::now();
	int result_cpu = (int)(end_cpu - start_cpu).count();
#endif

	// gpu
	hipMemcpy(d_a, a, sizeof(int) * SIZE, hipMemcpyHostToDevice);
	hipMemcpy(d_b, b, sizeof(int) * SIZE, hipMemcpyHostToDevice);

#if TIME_INSTANCE
	start_gpu = clock();
#else
	auto start_gpu = std::chrono::high_resolution_clock::now();
#endif
	kernel_add <<<SIZE, 1 >>> (d_a, d_b, d_c);
#if TIME_INSTANCE
	end_gpu = clock();
	result_gpu = (double)(end_gpu - start_gpu);
#else
	auto end_gpu = std::chrono::high_resolution_clock::now();
	int result_gpu = (int)(end_gpu - start_gpu).count();
#endif

	hipMemcpy(c, d_c, sizeof(int) * SIZE, hipMemcpyDeviceToHost);

#if DEBUG
	// 계산 결과를 출력합니다.
	std::cout << "a b c" << std::endl;
	std::cout << a << " " << b << " " << c << std::endl;
	//std::cout << *a << " " << *b << " " << *c << std::endl;
	//std::cout << &a << " " << &b << " " << &c << std::endl;
	for (int i = 0; i < SIZE; ++i)
		printf("c[%d] = %d\n", i, c[i]);
#endif

	// 계산 결과 확인
	for (int i = 0; i < SIZE; i++) {
		if (cpu_c[i] != c[i]) {
			std::cout << " 결과 부정확함 " << i << " 확인 필요 "  << std::endl;
			break;
		}
	}

	std::cout << "### 결과 정확함 ###" << std::endl;
	std::cout << "cpu 시간	gpu 시간" << std::endl;
	std::cout << result_cpu << "ns	" << result_gpu << "ns" << std::endl;
	std::cout << std::endl;
	std::cout << "gpu가 cpu보다 " << (result_cpu / result_gpu) << "배 빠름" << std::endl;

	delete[] a;
	delete[] b;
	delete[] c;

	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_c);

	system("pause");

	return 0;

}