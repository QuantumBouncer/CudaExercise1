#include "hip/hip_runtime.h"
// kernel �Լ� ȣ���ϱ�
// kernel �Լ��� ������ �����Ѵ�.
// blockIdx�� ����Ͽ� ���� ������ �����ϴ� ���� ��������.

#include ""
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
#include <iostream>

#define SIZE 100

__global__ void kernel_add(int *a, int *b, int *c) {
	//*c = *a + *b;
	// (###)
	c[blockIdx.x] = a[blockIdx.x] + b[blockIdx.x];
	printf("%d + %d = %d\n", a[blockIdx.x], b[blockIdx.x], c[blockIdx.x]);

}

int main(void) {
	// Host���� ����� �ν��Ͻ���
	//int a, b, c;
	// (###)
	int *a = new int[SIZE];
	int *b = new int[SIZE];
	int *c = new int[SIZE];

	// Device���� ����� �ν��Ͻ���
	int *d_a, *d_b, *d_c;

	// Device�� �޸𸮸� �Ҵ��մϴ�.
	//hipMalloc((void**)&d_a, sizeof(int));
	//hipMalloc((void**)&d_b, sizeof(int));
	//hipMalloc((void**)&d_c, sizeof(int));
	hipMalloc((void**)&d_a, sizeof(int) * SIZE);
	hipMalloc((void**)&d_b, sizeof(int) * SIZE);
	hipMalloc((void**)&d_c, sizeof(int) * SIZE);

	// �Է� ���� �����մϴ�.
	//a = 2;
	//b = 7;
	// (###)
	for (int i = 0; i < SIZE; i++) {
		a[i] = i;
		b[i] = i;
		std::cout << a[i] << " " << b[i] << std::endl;
	}


	// �Է� ���� Device memory ������ �����մϴ�.
	//hipMemcpy(d_a, &a, sizeof(int), hipMemcpyHostToDevice);
	//hipMemcpy(d_b, &b, sizeof(int), hipMemcpyHostToDevice);
	// (###)
	hipMemcpy(d_a, a, sizeof(int) * SIZE, hipMemcpyHostToDevice);
	hipMemcpy(d_b, b, sizeof(int) * SIZE, hipMemcpyHostToDevice);

	// kernel_add �Լ��� GPU���� �����մϴ�.
	//kernel_add <<< 1, 1 >>> (d_a, d_b, d_c);
	// ������ ���ڸ� 1,1 ���� N,1�� �ٲ㺾�ô�.
	//kernel_add <<< N, 1 >>> (d_a, d_b, d_c);
	// kernel_add �Լ��� N�� �����϶�� �ǹ��Դϴ�.
	// ���� N�� �����ϰ� �Ǹ� N�� ����� ������ �Լ��� ������ block�̶�� ������ ������ �˴ϴ�.
	// �� �Լ��� ����Ǵ� ������ grid ó�� �и��Ǿ� �ְ�, �̸� block��� �մϴ�.
	// �� �Լ� ���ο����� blockIdx.x�� �ڽ��� block ��ȣ�� Ȯ���� �� �ֽ��ϴ�.
	// �Լ��� ������ ���������� ȣ���Ͽ� ����ϱ� ���� �ڵ带 �ణ �����غ��ô�.(###)
	// (###)
	kernel_add << <SIZE, 1 >> > (d_a, d_b, d_c);
	
	// �Լ��� ����� Device ���� Host Memory�� �����մϴ�.
	//hipMemcpy(&c, d_c, sizeof(int), hipMemcpyDeviceToHost);
	// (###)
	hipMemcpy(c, d_c, sizeof(int) * SIZE, hipMemcpyDeviceToHost);

	// ��� ����� ����մϴ�.
	std::cout << "a b c" << std::endl;
	std::cout << a << " " << b << " " << c << std::endl;
	//std::cout << *a << " " << *b << " " << *c << std::endl;
	//std::cout << &a << " " << &b << " " << &c << std::endl;
	for (int i = 0; i < SIZE; ++i)
		printf("c[%d] = %d\n", i, c[i]);

	// (###)
	// cpu �޸𸮸� �����մϴ�.
	delete[] a;
	delete[] b;
	delete[] c;

	// gpu �޸𸮸� �����մϴ�.
	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_c);

	system("pause");

	return 0;

}