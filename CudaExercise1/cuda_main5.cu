#include "hip/hip_runtime.h"
// �̹��� 2���� ��� ���� ���� gpu������ �����ؼ� cpu ��� �ð��� �����غ���

#include ""
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
#include <iostream>

// ns ������ �ð� ������ ����
#include <chrono>

// matrix ����� ����
#include <vector>

using namespace std;

// cpu 
vector<vector<int>> MakeSquareMatrix(int matrix_size) {
	// 2���� ���� ����
	vector<vector<int>> a;
	for (int i = 0; i < matrix_size; i++) {
		vector<int> vector_1d(matrix_size);
		a.push_back(vector_1d);
	}

	// 2���� ���� �ʱ�ȭ
	for (int i = 0; i < matrix_size; i++) {
		for (int j = 0; j < matrix_size; j++) {
			a[i][j] = (j + 1) + (i * matrix_size);
		}
	}

	//// 2���� ���� ��� -> �Լ� ������
	//for (int i = 0; i < matrix_size; i++) {
	//	for (int j = 0; j < matrix_size; j++) {
	//		cout << a[i][j] << " ";
	//	}
	//	cout << endl;
	//}

	return a;
}

vector<vector<int>> MatrixMul(vector<vector<int>> matrix_A, vector<vector<int>> matrix_B) {			// �� ���� �Ẹ�ϱ� �����ͷ� �Է� �ִ� �ɷ� �ٲٰ� �ʹ�... �� �ٲ���
	vector<vector<int>> matrix_C;
	int matrix_size = matrix_A.size();
	for (int i = 0; i < matrix_size; i++) {
		vector<int> vector_1d(matrix_size);
		matrix_C.push_back(vector_1d);
	}

	int temp1, temp2;
	for (int i = 0; i < matrix_size; i++) {
		for (int j = 0; j < matrix_size; j++) {
			int sum = 0;
			for (int k = 0; k < matrix_size; k++) {
				sum += (matrix_A[i][k] * matrix_B[k][j]);
			}
			matrix_C[i][j] = sum;
		}
	}

	return matrix_C;
}

void PrintMatrix(vector<vector<int>> matrix) {
	// 2���� ���� ���
	int matrix_size = matrix.size();
	for (int i = 0; i < matrix_size; i++) {
		for (int j = 0; j < matrix_size; j++) {
			cout << matrix[i][j] << " ";
		}
		cout << endl;
	}
	cout << endl;
}



// gpu

__global__ vector<vector<int>> KernelMatrixMul(vector<vector<int>> matrix_A, vector<vector<int>> matrix_B) {

}


int main(void) {
	// cpu
	// �켱 cpu�� �̿��� ��� �� ���꿡 ���� �����մϴ�.
	// �ð��� �󸶳� �ɸ��� Ȯ���غ��ô�.
	int matrix_size = 7;
	vector<vector<int>> matrix_A;
	vector<vector<int>> matrix_B;
	vector<vector<int>> matrix_C;
	matrix_A = MakeSquareMatrix(matrix_size);
	matrix_B = MakeSquareMatrix(matrix_size);
	
	auto start_cpu = std::chrono::high_resolution_clock::now();
	matrix_C = MatrixMul(matrix_A, matrix_B);
	auto end_cpu = std::chrono::high_resolution_clock::now();
	int result_cpu = (int)(end_cpu - start_cpu).count();
	PrintMatrix(matrix_A);
	PrintMatrix(matrix_B);
	PrintMatrix(matrix_C);
	
	// gpu



	system("pause");
	return 0;
}