#include "hip/hip_runtime.h"
// kernel �Լ� ȣ���ϱ�
// kernel �Լ��� ���� �ƹ��͵� ����

#include ""
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
#include <iostream>
#define SIZE 1024

__global__ void mykernel(void) {
	// __global__�� �ǹ�
	// Device(GPU) ���� ����˴ϴ�.
	// Host(CPU) ���� ȣ��˴ϴ�.

	// Device Function(Device���� ����Ǵ� �Լ�)�� NVIDIA compiler�� ���� ó���˴ϴ�.
}

int main() {
	// Host Function(Host���� ����Ǵ� �Լ�)�� �Ϲ� compiler�� ���� ó���˴ϴ�.
	// <<< >>> �����ȣ 3���� Host�� Device�� ȣ���Ѵٰ� ��ŷ�ϴ� ���Դϴ�.
	// kernel launch ��� �մϴ�.
	// 
	mykernel << <1, 1 >> > ();
	std::cout << "Hello World!" << std::endl;

	system("pause");

	return 0;
}
