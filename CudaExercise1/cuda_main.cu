// cuda ���α׷��� ù ������ �ϱ� ���� �⺻ default �ڵ带 �����Դ�.
// ������ �����̴� �켱 c++��Ÿ�� �ڵ��̳� ������, SIZE ���ڸ� �ٲ㰡�� ������ ����� ���캸��
// ���� ���⼭ ��� ���� ���� cuda_main2.cu ���� �ϳ��� �������� ¤� ����


#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
#include <iostream>
#define SIZE 1024

// __global__�� ���ؼ� Ŀ������ ǥ���Ѵ�. host���� ȣ��ȴ�.
__global__ void VectorAdd(int *a, int *b, int *c, int n) {
	// ������ �����尡 ���ÿ� ó���Ѵ�.
	// ���� threadIdx(������ �ε���)�� ���ؼ� ��������� �����Ѵ�.
	int i = threadIdx.x;

	printf("threadIdx.x : %d, n : %d\n", i, n);

	for (i = 0; i < n; i++) {
		c[i] = a[i] + b[i];
		printf("%d = %d + %d\n", c[i], a[i], b[i]);
	}
}

int main() {
	//int *a, *b, *c;
	int *d_a, *d_b, *d_c;

	// ȣ��Ʈ�� �޸𸮿� �Ҵ��Ѵ�.
	// = CPU �޸𸮿� �Ҵ�
	//a = (int *)malloc(SIZE * sizeof(int));
	//b = (int *)malloc(SIZE * sizeof(int));
	//c = (int *)malloc(SIZE * sizeof(int));
	//int *a = new int[SIZE * sizeof(int)];
	//int *b = new int[SIZE * sizeof(int)];
	//int *c = new int[SIZE * sizeof(int)];
	int *a = new int[SIZE];
	int *b = new int[SIZE];
	int *c = new int[SIZE];

	//std::cout << a << " " << &a << " " << *a << std::endl;
	//std::cout << b << " " << &b << " " << *b << std::endl;
	//std::cout << c << " " << &c << " " << *c << std::endl;
	//system("pause");

	// cudaMalloc(destination, number of byte)�� device�� �޸𸮸� �Ҵ��Ѵ�.
	// = GPU �޸𸮿� �Ҵ�
	hipMalloc(&d_a, SIZE * sizeof(int));
	hipMalloc(&d_b, SIZE * sizeof(int));
	hipMalloc(&d_c, SIZE * sizeof(int)); 

	// �ʱ�ȭ
	for (int i = 0; i < SIZE; ++i)
	{
		a[i] = i;
		b[i] = 10;
		c[i] = 0;
	} 

	// cudaMemcpy(destination, source, number of byte, cudaMemcpyHostToDevice)�� ȣ��Ʈ���� ����̽��� �޸𸮸� ī���Ѵ�.
	hipMemcpy(d_a, a, SIZE * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_b, b, SIZE * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_c, c, SIZE * sizeof(int), hipMemcpyHostToDevice);

	// �Լ� ȣ���� ���ؼ� ���ο� ���ؽ� ��Ҹ� �߰��� �ʿ䰡 �ִ�.
	// ù��° parameter�� ���� ���̴�. ���������� ������ ���� �ϳ��̴�.
	// SIZE�� 1024���� �����带 �ǹ��Ѵ�.
	VectorAdd <<< 1, SIZE >>> (d_a, d_b, d_c, SIZE);

	//cudaMemcpy(source, destination, number of byte, cudaMemDeviceToHost)�� ����̽��� �޸�(���� ��� ������)�� ȣ��Ʈ�� ī���Ѵ�.
	hipMemcpy(a, d_a, SIZE * sizeof(int), hipMemcpyDeviceToHost);
	hipMemcpy(b, d_b, SIZE * sizeof(int), hipMemcpyDeviceToHost);
	hipMemcpy(c, d_c, SIZE * sizeof(int), hipMemcpyDeviceToHost);
	for (int i = 0; i < SIZE; ++i)
		printf("c[%d] = %d\n", i, c[i]);

	// ȣ��Ʈ�� �޸� �Ҵ� ����
	free(a);
	free(b);
	free(c);
	//delete[] a;
	//delete[] b;
	//delete[] c;

	// cudaFree(d_a)�� ���� ����̽��� �޸𸮸� �Ҵ� ����
	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_c);

	system("pause");

	return 0;
}